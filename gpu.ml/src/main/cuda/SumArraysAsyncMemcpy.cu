#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <ctime>

#define CheckErrorUtil(err) CheckError(err, __FUNCTION__, __LINE__)
#define CheckErrorMsgUtil(err, msg) CheckErrorMsg(err, msg, __FUNCTION__, __LINE__)

inline void CheckError(hipError_t const err, char const* const fun, const int line)
{
    if (err)
    {
        printf("CUDA Error Code[%d]: %s\n%s() Line:%d\n", err, hipGetErrorString(err), fun, line);
        exit(1);
    }
}

inline void CheckErrorMsg(hipError_t const err, char const* const msg, char const* const fun, int const line)
{
    if (err)
    {
        printf("CUDA Error Code[%d]: %s\n%s() Line:%d\n%s\n", err, hipGetErrorString(err), fun, line, msg);
        exit(1);
    }
}

void GenerateTestArrays(int const N, float* const a, float* const b, float* const c, float* const ref);
void CompareArrays(int const N, float const* const a, float const* const b);

__global__ void ArraysSum(float* const a, float* const b, float* const c, int const N)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i < N)
        c[i] = a[i] + b[i];
}

void GenerateTestArrays(int const N, float* const a, float* const b, float* const c, float* const ref)
{
    int i;

    srand((unsigned)time(NULL));

    for(i = 0; i < N; i++)
    {
        a[i] = (float)rand();
        b[i] = (float)rand();
        c[i] = 0.0f;
        ref[i] = a[i] + b[i];
    }
}

void CompareArrays(int const N, float const* const a, float const* const b)
{
    int i;
    int different = 0;

    for(i = 0; i < N; i++)
    {
        different = (a[i] != b[i]);
        if(different)
            break;
    }
    if(different)
    {
        printf("\n\n*** Oh No!  The GPU failed to sum the arrays. ***\n\n\n");
    }
    else
    {
        printf("\n\n*** Awesome!  The GPU summed the arrays!! ***\n\n\n");
    }
}

int main()
{
    std::clock_t start;
    start = std::clock();

    dim3 gridSize;
    dim3 blockSize;

    int    const N       = 4096000;
    size_t const N_BYTES = N * sizeof(float);
    int const BLOCK_SIZE = 512;

    float *aH, *bH, *cH, *refH;
    float *aD, *bD, *cD;

    aH = (float*)malloc(N_BYTES);
    bH = (float*)malloc(N_BYTES);
    cH = (float*)malloc(N_BYTES);
    refH = (float*)malloc(N_BYTES);

    printf("\n\nGenerating 2 random float arrays on Host - each of size %lu bytes...\n", N_BYTES);
    GenerateTestArrays(N, aH, bH, cH, refH);

    printf("Allocating %lu bytes on Device GPU to store the 2 generated arrays...\n", 2 * N_BYTES);
    CheckErrorUtil(hipMalloc((void**)&aD, N_BYTES));
    CheckErrorUtil(hipMalloc((void**)&bD, N_BYTES));

    printf("Allocating %lu bytes on Device GPU to store the result array after summing the 2 arrays...\n", N_BYTES);
    CheckErrorUtil(hipMalloc((void**)&cD, N_BYTES));

    // Use CUDA streams to manage the concurrency of copying and executing
    hipStream_t stream;
    hipStreamCreate(&stream);

    printf("Copying 2 arrays from Host to Device GPU using Streams...\n");
    hipMemcpyAsync(aD, aH, N_BYTES, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(bD, bH, N_BYTES, hipMemcpyHostToDevice, stream);

    blockSize.x = BLOCK_SIZE; blockSize.y = 1; blockSize.z = 1;
    gridSize.x = ((N + BLOCK_SIZE - 1) / BLOCK_SIZE); gridSize.y = 1; gridSize.z = 1;

    printf("Summing the 2 arrays and storing the result array on Device GPU...\n");
    ArraysSum<<<gridSize, blockSize>>>(aD, bD, cD, N);

    printf("Synchronizing the Device GPU memory before copying the result array back to Host...\n");
    CheckErrorUtil(hipStreamSynchronize(stream));

    printf("Copying result array from Device GPU to Host...\n");
    CheckErrorUtil(hipMemcpyAsync(cH, cD, N_BYTES, hipMemcpyDeviceToHost, stream));

    printf("Comparing expected result array stored on Host with actual result calculated on Device GPU...\n");
    CompareArrays(N, cH, refH);

    printf("Freeing %lu bytes on Device GPU...\n", 3 * N_BYTES);
    CheckErrorUtil(hipFree(aD));
    CheckErrorUtil(hipFree(bD));
    CheckErrorUtil(hipFree(cD));

    printf("Freeing memory on Host...\n");
    free(aH); 
    free(bH); 
    free(cH); 
    free(refH);

    printf("Resetting Device GPU as though nothing ever happened!\n\n");
    hipDeviceReset();

    printf("Executed in %.f milliseconds.\n\n", (std::clock() - start) / (double)(CLOCKS_PER_SEC / 1000));
    
    return 0;
}
